#include "hip/hip_runtime.h"
/*************************************************************************************/
/********************continous wavelet transform**************************************/
/* This is the main program for the CWT calculation, it uses fft technique to compute
the cwt coefficients. 

Author: Manas Jyoti Das, July:02:2016 */

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "cv.h"
#include "highgui.h"
#include<hipfft/hipfft.h>
#include "filter.cuh"
#include "normfilter.cuh"
#include<sys/time.h>

float *filter_dx,*filter_dy;

int main(void)
{
	IplImage* img=cvLoadImage("sample.jpg",CV_LOAD_IMAGE_COLOR);
	IplImage* gray_img=cvCreateImage(cvGetSize(img),IPL_DEPTH_8U,1);
	
	int i,temp=0,row,col,count=0;
	unsigned int img_dim;
	hipfftReal *signal;
	float *filter_dx_rearrange,*filter_dy_rearrange,*inverse_dx,*inverse_dy;
	hipfftReal *d_signal,*d_result;
	struct timeval t0,t1;
	long double elapsed;
	hipfftComplex *d_inverse,*mult_filt_data,*data_inverse;
	float scale[16];
	float temp1;
	
	img_dim=gray_img->width*gray_img->height;

	hipfftHandle plan_c2r,plan_r2c;

	cvCvtColor(img,gray_img,CV_RGB2GRAY);

	gettimeofday(&t0,0);	

	filter_dx=(float*)malloc(sizeof(float)*img_dim);
	filter_dy=(float*)malloc(sizeof(float)*img_dim);
	filter_dx_rearrange=(float*)malloc(sizeof(float)*(gray_img->height*(gray_img->width/2+1)));	
	filter_dy_rearrange=(float*)malloc(sizeof(float)*(gray_img->height*(gray_img->width/2+1)));	
	mult_filt_data=(hipfftComplex*)malloc(sizeof(hipfftComplex)*(gray_img->height*(gray_img->width/2+1)));
	inverse_dx=(float*)malloc(sizeof(float)*img_dim);
	inverse_dy=(float*)malloc(sizeof(float)*img_dim);
	data_inverse=(hipfftComplex*)malloc(sizeof(hipfftComplex)*(gray_img->height*(gray_img->width/2+1)));

	signal=(hipfftReal*)malloc(sizeof(hipfftReal)*img_dim);
	
	hipMalloc((void**)&d_signal,sizeof(hipfftReal)*img_dim);
	hipMalloc((void**)&d_result,sizeof(hipfftReal)*img_dim);
	hipMalloc((void**)&d_inverse,sizeof(hipfftComplex)*(gray_img->width)*(gray_img->height/2+1));
	
	for(temp1=1;temp1<4.1;temp1+=0.2)
	{
		scale[count]=pow(2,temp1); printf("scale is %f",scale[count]);
		count++;
	}
	count=0;
	hipfftPlan2d(&plan_c2r,gray_img->width,gray_img->height,HIPFFT_C2R);	
	hipfftPlan2d(&plan_r2c,gray_img->width,gray_img->height,HIPFFT_R2C);	

	for(row=0;row<gray_img->height;row++)
	{
		const uchar* ptr=(const uchar*)(gray_img->imageData+row*gray_img->widthStep);
		for(col=0;col<gray_img->width;col++)
		{
			signal[count]=*ptr++;
			count++;
		}
	}

	hipMemcpy(d_signal,signal,sizeof(hipfftReal)*img_dim,hipMemcpyHostToDevice);	

	hipfftExecR2C(plan_r2c, d_signal,d_inverse);

	hipMemcpy(data_inverse,d_inverse,sizeof(hipfftComplex)*(gray_img->width)*(gray_img->height/2+1),hipMemcpyDeviceToHost);
	
	for(temp=0;temp<16;temp++)
	{

	filter(filter_dx,filter_dy,gray_img->height,gray_img->width,scale[temp]);	

	memcpy(filter_dx_rearrange,filter_dx,sizeof(int)*(gray_img->width/2+1));
	memcpy(filter_dy_rearrange,filter_dy,sizeof(int)*(gray_img->width/2+1));

	for(i=1;i<=(gray_img->height-1);i++)
	{
		memcpy(filter_dx_rearrange+((i-1)*(gray_img->width/2+1)+(gray_img->width/2+1)),filter_dx+i*gray_img->width,sizeof(int)*(gray_img->width/2+1));
		memcpy(filter_dy_rearrange+((i-1)*(gray_img->width/2+1)+(gray_img->width/2+1)),filter_dy+i*gray_img->width,sizeof(int)*(gray_img->width/2+1));
	}

	//********** filter dx multiplication with data start *********//
	
	for(i=0;i<(gray_img->height*(gray_img->width/2+1));i++)
	{
		mult_filt_data[i].x= -filter_dx_rearrange[i]*data_inverse[i].y;
		mult_filt_data[i].y= filter_dx_rearrange[i]*data_inverse[i].x;
	}

	hipMemcpy(d_inverse,mult_filt_data,sizeof(hipfftComplex)*(gray_img->height*(gray_img->width/2+1)),hipMemcpyHostToDevice);

	hipfftExecC2R(plan_c2r,d_inverse,d_signal);

	hipMemcpy(signal,d_signal,sizeof(hipfftReal)*img_dim,hipMemcpyDeviceToHost);

	for(i=0;i<img_dim;i++)
	{
		inverse_dx[i]=signal[i]/img_dim;
	}
	//********** filter dx multiplication with data over *********//

	//********** filter dy multiplication with data start *********//

	for(i=0;i<(gray_img->height*(gray_img->width/2+1));i++)
	{
		mult_filt_data[i].x= -filter_dy_rearrange[i]*data_inverse[i].y;
		mult_filt_data[i].y= filter_dy_rearrange[i]*data_inverse[i].x;
	}

	hipMemcpy(d_inverse,mult_filt_data,sizeof(hipfftComplex)*(gray_img->height*(gray_img->width/2+1)),hipMemcpyHostToDevice);

	hipfftExecC2R(plan_c2r,d_inverse,d_signal);

	hipMemcpy(signal,d_signal,sizeof(hipfftReal)*img_dim,hipMemcpyDeviceToHost);

	for(i=0;i<img_dim;i++)
	{
		inverse_dy[i]=signal[i]/img_dim;
	}
	//********** filter dx multiplication with data over *********//

	//reusing filter_dx to hold magnitude and filter_dy to hold the angle, or i can use aliasing which is a good idea//
	for(i=0;i<img_dim;i++)
	{
		signal[i]=sqrt(pow(abs(inverse_dx[i]),2)+pow(abs(inverse_dy[i]),2));
	}
	
printf(" the complex number is %fi\n",signal[0]);

	}

	gettimeofday(&t1, 0);
	elapsed = (t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec;
	printf("\n time wall %Lf\n",elapsed/1000000);
	//free(filter_dx);
	//free(filter_dy);
	
	//cvReleaseImage(&img);
	//cvReleaseImage(&gray_img);
	
	return 0;
	
}
