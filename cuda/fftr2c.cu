#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>


#include<hipfft/hipfft.h>

#include "cv.h"
#include "highgui.h"

int main()
{
	int i,row,col,count=0;;
	hipfftReal *signal;
	hipfftReal *d_signal,*d_result;
	hipfftComplex *d_inverse;
	hipfftHandle plan_c2r,plan_r2c;	
	unsigned int img_dim;

	IplImage* img=cvLoadImage("sample.jpg",CV_LOAD_IMAGE_COLOR);
	IplImage* gray_img=cvCreateImage(cvGetSize(img),IPL_DEPTH_8U,1);

	img_dim=gray_img->width*gray_img->height;
	cvCvtColor(img,gray_img,CV_RGB2GRAY);

	signal=(hipfftReal*)malloc(sizeof(hipfftReal)*img_dim);

	for(row=0;row<gray_img->height;row++)
	{
		const uchar* ptr=(const uchar*)(gray_img->imageData+row*gray_img->widthStep);
		for(col=0;col<gray_img->width;col++)
		{
			signal[count]=*ptr++;
			count++;
		}
	}

	hipMalloc((void**)&d_signal,sizeof(hipfftReal)*img_dim);
	hipMalloc((void**)&d_result,sizeof(hipfftReal)*img_dim);
	hipMalloc((void**)&d_inverse,sizeof(hipfftComplex)*(gray_img->width)*(gray_img->height/2+1));

	//inverse=(hipfftComplex*)malloc(sizeof(hipfftComplex)*NUM_POINTS);

	hipfftPlan2d(&plan_c2r,gray_img->width,gray_img->height,HIPFFT_C2R);	
	hipfftPlan2d(&plan_r2c,gray_img->width,gray_img->height,HIPFFT_R2C);

	hipMemcpy(d_signal,signal,sizeof(hipfftReal)*img_dim,hipMemcpyHostToDevice);

	hipfftExecR2C(plan_r2c, d_signal,d_inverse);
	
	//forward_fft<<<NUM_POINTS,1>>>(d_signal,d_inverse);

	//hipMemcpy(inverse,d_inverse,(hipfftComplex)*NUM_POINTS,hipMemcpyDeviceToHost);

	for(i=0;i<64;i++)
		printf("[%f]",signal[i]);

	hipfftExecC2R(plan_c2r,d_inverse,d_result);

	hipMemcpy(signal,d_result,sizeof(hipfftReal)*img_dim,hipMemcpyDeviceToHost);	
printf("\n______________________________________________\n");
		
	for(i=0;i<64;i++)
		printf("[%f]",signal[i]/img_dim);	

	hipfftDestroy(plan_c2r);
	hipfftDestroy(plan_r2c);
	hipFree(d_signal);
	hipFree(d_result);
	hipFree(d_inverse);
	hipFree(signal);
	//free(inverse);
	return 0;
}
